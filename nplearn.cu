#include "hip/hip_runtime.h"
/*
CUDANPLEARN General kernel support functions
Copyright (C) 2013 Zhouyisu <zhouyisu # gmail.com>

This program is free software; you can redistribute it and/or
modify it under the terms of the GNU General Public License
as published by the Free Software Foundation; either version 2
of the License, or (at your option) any later version.

This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.

You should have received a copy of the GNU General Public License
along with this program; if not, write to the Free Software
Foundation, Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301, USA.
*/

#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "nplearn.cuh"
#include "npconstant.cuh"
#include <string.h>
#include <algorithm>
#include <utility>
#include <stdio.h>

inline bool checkCUDAError() {
    hipError_t err = hipGetLastError();
    return hipSuccess != err;
}

template<typename T>
inline T&min(const T&a,const T&b)
{return a<b?a:b;}


extern "C" {
	
	/*ToHidden Kernel ˵��

	���㲢�з�ʽ���������鲢��
	dataΪ: {y����(cy), {x����(cx), {ͨ��(cc), {������(d)}}}} �洢�ṹ,  dΪ16����
	weightΪ: {���ͨ��(wc), {���y����(wyo), {���x����(wxo), {����y����(wyp), {����x����(wxp), {����ͨ��(wop)}}}}}} �洢�ṹ ( cc==wop )
	outΪ�� {ͨ��(wc), {y����(wyo), {x����(wxo), {������(d)}}}} �洢�ṹ

	ʵ�֣�
	data�ֳ���Ϊ 32*7*n�ĶΣ����Ұ���Kernel����ķ�ʽ����
	weightֱ�Ӵ洢���Դ���
	��δ���data����С��100M����ӽ��ķֽⷽ�����зֶ�
	 */
	DLL extern int tohidden_kern(float*data, float*weight, float*out, int d, int cy, int cx, int cc, int wc, int wyo, int wxo, int wyp, int wxp) {
		setconstant(cy,cx,cc,wc,wyo,wxo,wyp,wxp);
		int blocksize = cy*cx*cc*sizeof(float);
		int outblocksize = wc*wyo*wxo*sizeof(float);

		int datablock = min(CUDAMemory / blocksize, CUDAMemory / outblocksize);
		if (datablock>d) datablock=d+CUDAAligns-1;
		datablock = datablock/CUDAAligns*CUDAAligns;
		if (datablock>5600) datablock=5600;
		if (datablock==0) return INSUFFICIENT_MEMORY;
		int outsize = datablock*wc*wyo*wxo*sizeof(float);

		//Initial devicedata, deviceweight, deviceout, hostout
		float*hostdata,*devicedata,*deviceweight,*deviceout,*hostout;
		hostout=(float*)malloc(outsize);
		hostdata=(float*)malloc(datablock*blocksize);

		CUDAASSERT(hipMalloc(&deviceweight,wc*wyo*wxo*wyp*wxp*cc*sizeof(float)));
		CUDAASSERT(hipMemcpy(deviceweight,weight,wc*wyo*wxo*wyp*wxp*cc*sizeof(float),hipMemcpyHostToDevice));
		CUDAASSERT(hipMalloc(&devicedata,datablock*blocksize));
		CUDAASSERT(hipMalloc(&deviceout,outsize));

		for (unsigned long long pd=0;pd<d;pd+=datablock) {
			//Prepare hostdata
			int pdo=0;
			for (;pdo<datablock;pdo++) if (pdo+pd<d)
				for (int py=0;py<cy;py++)
					for (int px=0;px<cx;px++)
						for (int pc=0;pc<cc;pc++)
							hostdata[((py*cx+px)*cc+pc)*datablock+pdo]=data[(((pd+pdo)*cy+py)*cx+px)*cc+pc];
			else break;

			dim3 gridDim;
			long xval = (datablock*wc*wyo*wxo+CUDALines-1)/CUDALines;
			gridDim.y=(xval+1023)/1024; if (xval>=1024) gridDim.x=1024; else gridDim.x=xval;
			gridDim.z=(gridDim.y+1023)/1024; if (gridDim.y>=1024) gridDim.y=1024;
			CUDAASSERT(hipMemcpy(devicedata,hostdata,datablock*blocksize,hipMemcpyHostToDevice));
			tohidden_dataparallel<<<gridDim,CUDALines>>>(devicedata,deviceweight,deviceout,datablock,pdo);
			CUDAASSERT(hipMemcpy(hostout,deviceout,outsize,hipMemcpyDeviceToHost));

			for (pdo=0;pdo<datablock;pdo++) if (pdo+pd<d)
				for (int pc=0;pc<wc;pc++)
					for (int py=0;py<wyo;py++)
						for (int px=0;px<wxo;px++)
							out[(((pd+pdo)*wc+pc)*wyo+py)*wxo+px]=hostout[((pc*wyo+py)*wxo+px)*datablock+pdo];
			else break;
		}
		//Release memory associated
		CUDAASSERT(hipFree(devicedata));
		CUDAASSERT(hipFree(deviceweight));
		CUDAASSERT(hipFree(deviceout));
		free(hostout);
		free(hostdata);
		return 0;
	}
	/*FromHidden Kernel ˵��

	���㲢�з�ʽ���������鲢��
	hiddenΪ: {ͨ��(wc), {y����(wyo), {x����(wxo), {������(d)}}}} �洢�ṹ,  dΪ16����
	weightΪ: {���ͨ��(wc), {���y����(wyo), {���x����(wxo), {����y����(wyp), {����x����(wxp), {����ͨ��(wop)}}}}}} �洢�ṹ ( cc==wop )
	outΪ�� {y����(cy), {x����(cx), {ͨ��(cc), {������(d)}}}} �洢�ṹ

	ʵ�֣�
	hidden�ֳ���Ϊ 32*7*n�ĶΣ����Ұ���Kernel����ķ�ʽ����
	weightֱ�Ӵ洢���Դ���
	��δ���data����С��100M����ӽ��ķֽⷽ�����зֶ�
	 */
	DLL extern int fromhidden_kern(float*hidden, float*weight, float*dataout, int d, int cy, int cx, int cc, int wc, int wyo, int wxo, int wyp, int wxp) {
		setconstant(cy,cx,cc,wc,wyo,wxo,wyp,wxp);
		int blocksize = cy*cx*cc*sizeof(float);
		int outblocksize = wc*wyo*wxo*sizeof(float);

		int datablock = min(CUDAMemory / blocksize, CUDAMemory / outblocksize);
		if (datablock>d) datablock=d+CUDAAligns-1;
		datablock = datablock/CUDAAligns*CUDAAligns;
		if (datablock>5600) datablock=5600;
		if (datablock==0) return INSUFFICIENT_MEMORY;
		int outsize = datablock*wc*wyo*wxo*sizeof(float);

		float*hostdata,*devicedata,*deviceweight,*deviceout,*hostout;
		CUDAASSERT(hipMalloc(&deviceweight,wc*wyo*wxo*wyp*wxp*cc*sizeof(float)));
		CUDAASSERT(hipMemcpy(deviceweight,weight,wc*wyo*wxo*wyp*wxp*cc*sizeof(float),hipMemcpyHostToDevice));
		CUDAASSERT(hipMalloc(&deviceout,outsize));
		CUDAASSERT(hipMalloc(&devicedata,datablock*cy*cx*cc*sizeof(float)));
		hostout=(float*)malloc(outsize);
		hostdata=(float*)malloc(datablock*blocksize);

		for (unsigned long long pd=0;pd<d;pd+=datablock) {
			int pdo=0;
			for (;pdo<datablock;pdo++) if (pdo+pd<d)
				for (int pc=0;pc<wc;pc++)
					for (int py=0;py<wyo;py++)
						for (int px=0;px<wxo;px++)
							hostout[((pc*wyo+py)*wxo+px)*datablock+pdo]=hidden[(((pd+pdo)*wc+pc)*wyo+py)*wxo+px];
			else break;

			dim3 gridDim;
			gridDim.x=(datablock*cy*cx*cc+CUDALines-1)/CUDALines;
			gridDim.y=(gridDim.x+1023)/1024; if (gridDim.x>=1024) gridDim.x=1024;
			gridDim.z=(gridDim.y+1023)/1024; if (gridDim.y>=1024) gridDim.y=1024;

			CUDAASSERT(hipMemcpy(deviceout,hostout,outsize,hipMemcpyHostToDevice));
			fromhidden2_dataparallel<<<gridDim,CUDALines>>>(deviceout,deviceweight,devicedata,datablock,pdo);
			CUDAASSERT(hipMemcpy(hostdata,devicedata,datablock*blocksize,hipMemcpyDeviceToHost));

			for (pdo=0;pdo<datablock;pdo++) if (pdo+pd<d)
				for (int py=0;py<cy;py++)
					for (int px=0;px<cx;px++)
						for (int pc=0;pc<cc;pc++)
							dataout[(((pd+pdo)*cy+py)*cx+px)*cc+pc]=hostdata[((py*cx+px)*cc+pc)*datablock+pdo];
			else break;
		}
		CUDAASSERT(hipFree(devicedata));
		CUDAASSERT(hipFree(deviceweight));
		CUDAASSERT(hipFree(deviceout));
		free(hostout);
		free(hostdata);
		return 0;
	}
	/*ExtractValue Kernel ˵��

	���㲢�з�ʽ���������ڹ㷺���� wxp*wop 32��������λ��blockIdx.x����������λ��blockIdx.y��blockIdx.z

	dataΪ�� {������(d), {y����(cy), {x����(cx), {ͨ��(cc)}}}} �洢�ṹ
	hiddenΪ: {������(d), {ͨ��(wc), {y����(wyo), {x����(wxo)}}}} �洢�ṹ
	dweightΪ: {���ͨ��(wc), {���y����(wyo), {���x����(wxo), {����y����(wyp), {����x����(wxp), {����ͨ��(wop)}}}}}} �洢�ṹ ( cc==wop )

	ʵ�֣�
	����data���100M�Ķη������ڴ���
	weight���Դ����ۻ�
	ÿ��thread�������weight��һ��ֵ
	 */
	DLL extern int extractvalue_kern(float*data, float*hidden, float*weight, int d, int cy, int cx, int cc, int wc, int wyo, int wxo, int wyp, int wxp) {
		setconstant(cy,cx,cc,wc,wyo,wxo,wyp,wxp);
		int blocksize = cy*cx*cc*sizeof(float);
		int outblocksize = wc*wyo*wxo*sizeof(float);

		int datablock = min(CUDAMemory / blocksize, CUDAMemory / outblocksize);
		if (datablock>d) datablock=d;
		if (datablock>5120) datablock=5120;
		if (datablock==0) return INSUFFICIENT_MEMORY;
		int splits = wxo*wxp*cc;
		if (splits>1024) splits=1024;
		
		//Initial devicedata, deviceweight, deviceout, hostout
		float*devicedata,*deviceweight,*devicehidden;
		CUDAASSERT(hipMalloc(&deviceweight,wc*wyo*wxo*wyp*wxp*cc*sizeof(float)));
		CUDAASSERT(hipMemset(deviceweight,0,wc*wyo*wxo*wyp*wxp*cc*sizeof(float)));
		CUDAASSERT(hipMalloc(&devicedata,datablock*blocksize));
		CUDAASSERT(hipMalloc(&devicehidden,datablock*outblocksize));

		for (unsigned long long pd=0;pd<d;pd+=datablock) {
			int sz=datablock;
			if (pd+datablock>=d) sz=d-pd;

			int totaldim=wc*wyo*wxo*wyp*wxp*cc;
			dim3 gridDim;
			gridDim.x=(totaldim+splits-1)/splits;
			gridDim.y=(gridDim.x+1023)/1024; if (gridDim.x>=1024) gridDim.x=1024;
			gridDim.z=(gridDim.y+1023)/1024; if (gridDim.y>=1024) gridDim.y=1024;

			CUDAASSERT(hipMemcpy(devicedata,data+pd*cy*cx*cc,sz*blocksize,hipMemcpyHostToDevice));
			CUDAASSERT(hipMemcpy(devicehidden,hidden+pd*wc*wyo*wxo,sz*outblocksize,hipMemcpyHostToDevice));
			extractvalue_dataparallel<<<gridDim,splits>>>(devicedata,devicehidden,deviceweight,sz);
		}
		//Copyout result
		CUDAASSERT(hipMemcpy(weight,deviceweight,wc*wyo*wxo*wyp*wxp*cc*sizeof(float),hipMemcpyDeviceToHost));
		//Release memory associated
		CUDAASSERT(hipFree(devicedata));
		CUDAASSERT(hipFree(deviceweight));
		CUDAASSERT(hipFree(devicehidden));
		return 0;
	}
	DLL extern int pooling_in_kern(float*hiddenin, float*poolingout, int d, int wc, int wyo, int wxo, int poolsize) //����ֻȡ��Ч��Χ��pooling����
	{
		int blocksize = wc*wyo*wxo*sizeof(float);
		int cyo=wyo-poolsize+1;
		int cxo=wxo-poolsize+1;
		int poolblocksize = wc*cyo*cxo*sizeof(float);
		
		int datablock = CUDAMemory / blocksize;
		if (datablock>d) datablock=d;
		if (datablock==0) return INSUFFICIENT_MEMORY;
		
		float*devicehidden,*devicepooling;
		CUDAASSERT(hipMalloc(&devicehidden,datablock*blocksize));
		CUDAASSERT(hipMalloc(&devicepooling,datablock*poolblocksize));
		
		for (unsigned long long pd=0;pd<d;pd+=datablock) {
			CUDAASSERT(hipMemcpy(devicehidden,hiddenin+(wc*wyo*wxo*pd),(pd+datablock<=d)?datablock*blocksize:(d-pd)*blocksize,hipMemcpyHostToDevice));
			pooling_in_dataparallel<<<(cyo*cxo+CUDALines-1)/CUDALines,CUDALines>>>(devicehidden,devicepooling,(pd+datablock<=d)?datablock:(d-pd),wc,wyo,wxo,poolsize);
			CUDAASSERT(hipMemcpy(poolingout+(wc*cyo*cxo*pd),devicepooling,(pd+datablock<=d)?datablock*poolblocksize:(d-pd)*poolblocksize,hipMemcpyDeviceToHost));
		}

		CUDAASSERT(hipFree(devicehidden));
		CUDAASSERT(hipFree(devicepooling));
		return 0;
	}
	DLL extern int pooling_out_kern(float*poolingin, float*hiddenout, int d, int wc, int wyo, int wxo, int poolsize) //���������������pooling����
	{
		int blocksize = wc*wyo*wxo*sizeof(float);
		int cyo=wyo-poolsize+1;
		int cxo=wxo-poolsize+1;
		int poolblocksize = wc*cyo*cxo*sizeof(float);
		
		int datablock = CUDAMemory / blocksize;
		if (datablock>d) datablock=d;
		if (datablock==0) return INSUFFICIENT_MEMORY;
		float*devicehidden,*devicepooling;

		CUDAASSERT(hipMalloc(&devicehidden,datablock*blocksize));
		CUDAASSERT(hipMalloc(&devicepooling,datablock*poolblocksize));
		
		for (unsigned long long pd=0;pd<d;pd+=datablock) {
			CUDAASSERT(hipMemcpy(devicepooling,poolingin+(wc*cyo*cxo*pd),((pd+datablock<=d)?datablock*poolblocksize:(d-pd)*poolblocksize),hipMemcpyHostToDevice));
			CUDAASSERT(hipMemset(devicehidden,0,datablock*blocksize));
			pooling_back_dataparallel<<<(cyo*cxo+CUDALines-1)/CUDALines,CUDALines>>>(devicepooling,devicehidden,(pd+datablock<=d)?datablock:(d-pd),wc,wyo,wxo,poolsize);
			CUDAASSERT(hipMemcpy(hiddenout+(wc*wyo*wxo*pd),devicehidden,((pd+datablock<=d)?datablock*blocksize:(d-pd)*blocksize),hipMemcpyDeviceToHost));
		}

		CUDAASSERT(hipFree(devicehidden));
		CUDAASSERT(hipFree(devicepooling));
		return 0;
	}
	DLL extern void layerpooling_in(float*__restrict hiddenin, float*__restrict poolingout, int d, int wc, int wyo, int wxo, int poolsize)
	{
		int outl=wc/poolsize;
		memset(poolingout,0,d*outl*wyo*wxo*sizeof(float));
		int wn=wyo*wxo;
		for (unsigned long long pd=0;pd<d;pd++)
			for (int pp=0;pp<outl;pp++)
				for (int pp2=0;pp2<poolsize;pp2++)
					for (int pn=0;pn<wn;pn++)
							poolingout[(pd*outl+pp)*wn+pn]+=hiddenin[(pd*wc+pp*poolsize+pp2)*wn+pn];
	}
	DLL extern void layerpooling_out(float*__restrict poolingin, float*__restrict hiddenout, int d, int wc, int wyo, int wxo, int poolsize)
	{
		int outl=wc/poolsize;
		int wn=wyo*wxo;
		for (unsigned long long pd=0;pd<d;pd++)
			for (int pp=0;pp<outl;pp++)
				for (int pp2=0;pp2<poolsize;pp2++)
					for (int pn=0;pn<wn;pn++)
							hiddenout[(pd*wc+pp*poolsize+pp2)*wn+pn]=poolingin[(pd*outl+pp)*wn+pn];
	}

	DLL extern int inlayermax_keepshape(float*hiddenin, float*poolingout, int d, int dc, int dy, int dx, int poolsize)
	{
		int blocksize = dc*dy*dx*sizeof(float);		
		int datablock = CUDAMemory / blocksize;
		if (datablock>d) datablock=d;
		if (datablock==0) return INSUFFICIENT_MEMORY;
		
		float*devicehidden,*devicepooling;
		CUDAASSERT(hipMalloc(&devicehidden,datablock*blocksize));
		CUDAASSERT(hipMalloc(&devicepooling,datablock*blocksize));
		
		for (unsigned long long pd=0;pd<d;pd+=datablock) {
			CUDAASSERT(hipMemcpy(devicehidden,hiddenin+dc*dy*dx*pd,(pd+datablock<=d)?datablock*blocksize:(d-pd)*blocksize,hipMemcpyHostToDevice));
			kinlayermax<<<(dy*dx+CUDALines-1)/CUDALines,CUDALines>>>(devicehidden,devicepooling,(pd+datablock<=d)?datablock:(d-pd),dc,dy,dx,poolsize);
			CUDAASSERT(hipMemcpy(poolingout+dc*dy*dx*pd,devicepooling,(pd+datablock<=d)?datablock*blocksize:(d-pd)*blocksize,hipMemcpyDeviceToHost));
		}

		CUDAASSERT(hipFree(devicehidden));
		CUDAASSERT(hipFree(devicepooling));
		return 0;
	}
	//follow version of inlayermax is the same function as reverseinlayermax
	DLL extern int reverseinlayermax_keepshape(float*hiddenin, float*grad, float*outgrad, int d, int dc, int dy, int dx, int poolsize)
	{
		int blocksize = dc*dy*dx*sizeof(float);
		int datablock = CUDAMemory / blocksize;
		if (datablock>d) datablock=d;
		if (datablock==0) return INSUFFICIENT_MEMORY;
		float*devicehidden,*devicegrad,*devicehgrad;

		CUDAASSERT(hipMalloc(&devicehidden,datablock*blocksize));
		CUDAASSERT(hipMalloc(&devicegrad,datablock*blocksize));
		CUDAASSERT(hipMalloc(&devicehgrad,datablock*blocksize));
		
		for (unsigned long long pd=0;pd<d;pd+=datablock) {
			CUDAASSERT(hipMemcpy(devicehidden,hiddenin+(dc*dy*dx*pd),((pd+datablock<=d)?datablock*blocksize:(d-pd)*blocksize),hipMemcpyHostToDevice));
			CUDAASSERT(hipMemcpy(devicegrad,grad+(dc*dy*dx*pd),((pd+datablock<=d)?datablock*blocksize:(d-pd)*blocksize),hipMemcpyHostToDevice));
			kreverseinlayermax<<<(dy*dx+CUDALines-1)/CUDALines,CUDALines>>>(devicehidden,devicegrad,devicehgrad,(pd+datablock<=d)?datablock:(d-pd),dc,dy,dx,poolsize);
			CUDAASSERT(hipMemcpy(outgrad+(dc*dy*dx*pd),devicehgrad,((pd+datablock<=d)?datablock*blocksize:(d-pd)*blocksize),hipMemcpyDeviceToHost));
		}

		CUDAASSERT(hipFree(devicehidden));
		CUDAASSERT(hipFree(devicegrad));
		CUDAASSERT(hipFree(devicehgrad));
		return 0;
	}

	DLL extern int alllayermax_keepshape(float*hiddenin, float*poolingout, int d, int dc, int dy, int dx, int poolsize)
	{
		int blocksize = dc*dy*dx*sizeof(float);		
		int datablock = CUDAMemory / blocksize;
		if (datablock>d) datablock=d;
		if (datablock==0) return INSUFFICIENT_MEMORY;
		
		float*devicehidden,*devicepooling;
		CUDAASSERT(hipMalloc(&devicehidden,datablock*blocksize));
		CUDAASSERT(hipMalloc(&devicepooling,datablock*blocksize));
		
		for (unsigned long long pd=0;pd<d;pd+=datablock) {
			CUDAASSERT(hipMemcpy(devicehidden,hiddenin+dc*dy*dx*pd,(pd+datablock<=d)?datablock*blocksize:(d-pd)*blocksize,hipMemcpyHostToDevice));
			kalllayermax<<<(dy*dx*dc+CUDALines-1)/CUDALines,CUDALines>>>(devicehidden,devicepooling,(pd+datablock<=d)?datablock:(d-pd),dc,dy,dx,poolsize);
			CUDAASSERT(hipMemcpy(poolingout+dc*dy*dx*pd,devicepooling,(pd+datablock<=d)?datablock*blocksize:(d-pd)*blocksize,hipMemcpyDeviceToHost));
		}

		CUDAASSERT(hipFree(devicehidden));
		CUDAASSERT(hipFree(devicepooling));
		return 0;
	}

	//follow version of inlayermax is the same function as reverseinlayermax
	DLL extern int reversealllayermax_keepshape(float*hiddenin, float*grad, float*outgrad, int d, int dc, int dy, int dx, int poolsize)
	{
		int blocksize = dc*dy*dx*sizeof(float);
		int datablock = CUDAMemory / blocksize;
		if (datablock>d) datablock=d;
		if (datablock==0) return INSUFFICIENT_MEMORY;
		float*devicehidden,*devicegrad,*devicehgrad;

		CUDAASSERT(hipMalloc(&devicehidden,datablock*blocksize));
		CUDAASSERT(hipMalloc(&devicegrad,datablock*blocksize));
		CUDAASSERT(hipMalloc(&devicehgrad,datablock*blocksize));
		
		for (unsigned long long pd=0;pd<d;pd+=datablock) {
			CUDAASSERT(hipMemcpy(devicehidden,hiddenin+(dc*dy*dx*pd),((pd+datablock<=d)?datablock*blocksize:(d-pd)*blocksize),hipMemcpyHostToDevice));
			CUDAASSERT(hipMemcpy(devicegrad,grad+(dc*dy*dx*pd),((pd+datablock<=d)?datablock*blocksize:(d-pd)*blocksize),hipMemcpyHostToDevice));
			kreversealllayermax<<<(dy*dx*dc+CUDALines-1)/CUDALines,CUDALines>>>(devicehidden,devicegrad,devicehgrad,(pd+datablock<=d)?datablock:(d-pd),dc,dy,dx,poolsize);
			CUDAASSERT(hipMemcpy(outgrad+(dc*dy*dx*pd),devicehgrad,((pd+datablock<=d)?datablock*blocksize:(d-pd)*blocksize),hipMemcpyDeviceToHost));
		}

		CUDAASSERT(hipFree(devicehidden));
		CUDAASSERT(hipFree(devicegrad));
		CUDAASSERT(hipFree(devicehgrad));
		return 0;
	}

	DLL extern int inlayerabsmax_keepshape(float*hiddenin, float*poolingout, int d, int dc, int dy, int dx, int poolsize)
	{
		int blocksize = dc*dy*dx*sizeof(float);		
		int datablock = CUDAMemory / blocksize;
		if (datablock>d) datablock=d;
		if (datablock==0) return INSUFFICIENT_MEMORY;
		
		float*devicehidden,*devicepooling;
		CUDAASSERT(hipMalloc(&devicehidden,datablock*blocksize));
		CUDAASSERT(hipMalloc(&devicepooling,datablock*blocksize));
		
		for (unsigned long long pd=0;pd<d;pd+=datablock) {
			CUDAASSERT(hipMemcpy(devicehidden,hiddenin+dc*dy*dx*pd,(pd+datablock<=d)?datablock*blocksize:(d-pd)*blocksize,hipMemcpyHostToDevice));
			kinlayerabsmax<<<(dy*dx+CUDALines-1)/CUDALines,CUDALines>>>(devicehidden,devicepooling,(pd+datablock<=d)?datablock:(d-pd),dc,dy,dx,poolsize);
			CUDAASSERT(hipMemcpy(poolingout+dc*dy*dx*pd,devicepooling,(pd+datablock<=d)?datablock*blocksize:(d-pd)*blocksize,hipMemcpyDeviceToHost));
		}

		CUDAASSERT(hipFree(devicehidden));
		CUDAASSERT(hipFree(devicepooling));
		return 0;
	}
	//follow version of inlayermax is the same function as reverseinlayermax
	DLL extern int reverseinlayerabsmax_keepshape(float*hiddenin, float*grad, float*outgrad, int d, int dc, int dy, int dx, int poolsize)
	{
		int blocksize = dc*dy*dx*sizeof(float);
		int datablock = CUDAMemory / blocksize;
		if (datablock>d) datablock=d;
		if (datablock==0) return INSUFFICIENT_MEMORY;
		float*devicehidden,*devicegrad,*devicehgrad;

		CUDAASSERT(hipMalloc(&devicehidden,datablock*blocksize));
		CUDAASSERT(hipMalloc(&devicegrad,datablock*blocksize));
		CUDAASSERT(hipMalloc(&devicehgrad,datablock*blocksize));
		
		for (unsigned long long pd=0;pd<d;pd+=datablock) {
			CUDAASSERT(hipMemcpy(devicehidden,hiddenin+(dc*dy*dx*pd),((pd+datablock<=d)?datablock*blocksize:(d-pd)*blocksize),hipMemcpyHostToDevice));
			CUDAASSERT(hipMemcpy(devicegrad,grad+(dc*dy*dx*pd),((pd+datablock<=d)?datablock*blocksize:(d-pd)*blocksize),hipMemcpyHostToDevice));
			kreverseinlayerabsmax<<<(dy*dx+CUDALines-1)/CUDALines,CUDALines>>>(devicehidden,devicegrad,devicehgrad,(pd+datablock<=d)?datablock:(d-pd),dc,dy,dx,poolsize);
			CUDAASSERT(hipMemcpy(outgrad+(dc*dy*dx*pd),devicehgrad,((pd+datablock<=d)?datablock*blocksize:(d-pd)*blocksize),hipMemcpyDeviceToHost));
		}

		CUDAASSERT(hipFree(devicehidden));
		CUDAASSERT(hipFree(devicegrad));
		CUDAASSERT(hipFree(devicehgrad));
		return 0;
	}

	DLL extern int alllayerabsmax_keepshape(float*hiddenin, float*poolingout, int d, int dc, int dy, int dx, int poolsize)
	{
		int blocksize = dc*dy*dx*sizeof(float);		
		int datablock = CUDAMemory / blocksize;
		if (datablock>d) datablock=d;
		if (datablock==0) return INSUFFICIENT_MEMORY;
		
		float*devicehidden,*devicepooling;
		CUDAASSERT(hipMalloc(&devicehidden,datablock*blocksize));
		CUDAASSERT(hipMalloc(&devicepooling,datablock*blocksize));
		
		for (unsigned long long pd=0;pd<d;pd+=datablock) {
			CUDAASSERT(hipMemcpy(devicehidden,hiddenin+dc*dy*dx*pd,(pd+datablock<=d)?datablock*blocksize:(d-pd)*blocksize,hipMemcpyHostToDevice));
			kalllayerabsmax<<<(dy*dx*dc+CUDALines-1)/CUDALines,CUDALines>>>(devicehidden,devicepooling,(pd+datablock<=d)?datablock:(d-pd),dc,dy,dx,poolsize);
			CUDAASSERT(hipMemcpy(poolingout+dc*dy*dx*pd,devicepooling,(pd+datablock<=d)?datablock*blocksize:(d-pd)*blocksize,hipMemcpyDeviceToHost));
		}

		CUDAASSERT(hipFree(devicehidden));
		CUDAASSERT(hipFree(devicepooling));
		return 0;
	}

	//follow version of inlayermax is the same function as reverseinlayermax
	DLL extern int reversealllayerabsmax_keepshape(float*hiddenin, float*grad, float*outgrad, int d, int dc, int dy, int dx, int poolsize)
	{
		int blocksize = dc*dy*dx*sizeof(float);
		int datablock = CUDAMemory / blocksize;
		if (datablock>d) datablock=d;
		if (datablock==0) return INSUFFICIENT_MEMORY;
		float*devicehidden,*devicegrad,*devicehgrad;

		CUDAASSERT(hipMalloc(&devicehidden,datablock*blocksize));
		CUDAASSERT(hipMalloc(&devicegrad,datablock*blocksize));
		CUDAASSERT(hipMalloc(&devicehgrad,datablock*blocksize));
		
		for (unsigned long long pd=0;pd<d;pd+=datablock) {
			CUDAASSERT(hipMemcpy(devicehidden,hiddenin+(dc*dy*dx*pd),((pd+datablock<=d)?datablock*blocksize:(d-pd)*blocksize),hipMemcpyHostToDevice));
			CUDAASSERT(hipMemcpy(devicegrad,grad+(dc*dy*dx*pd),((pd+datablock<=d)?datablock*blocksize:(d-pd)*blocksize),hipMemcpyHostToDevice));
			kreversealllayerabsmax<<<(dy*dx*dc+CUDALines-1)/CUDALines,CUDALines>>>(devicehidden,devicegrad,devicehgrad,(pd+datablock<=d)?datablock:(d-pd),dc,dy,dx,poolsize);
			CUDAASSERT(hipMemcpy(outgrad+(dc*dy*dx*pd),devicehgrad,((pd+datablock<=d)?datablock*blocksize:(d-pd)*blocksize),hipMemcpyDeviceToHost));
		}

		CUDAASSERT(hipFree(devicehidden));
		CUDAASSERT(hipFree(devicegrad));
		CUDAASSERT(hipFree(devicehgrad));
		return 0;
	}

	/*
	This function does a max block pooling
	data is [nd [dc [dy [dx [float]]]]]
	out is [nd [dc [(dy+size-1)/size [(dx+size-1)/size [float]]]]]
	*/
	DLL extern int maxblock2D(float*data,float*out,int nd,int dc,int dy,int dx,int size)
	{
		int oy=(dy+size-1)/size,ox=(dx+size-1)/size;

		int datablocksize=dc*dy*dx*sizeof(float);
		int outblocksize=dc*oy*ox*sizeof(float);
		int blockcount=CUDAMemory/datablocksize;
		if (blockcount>nd) blockcount=nd+CUDAAligns-1;
		blockcount=blockcount/CUDAAligns*CUDAAligns;
		if (blockcount==0) return INSUFFICIENT_MEMORY;

		float*hostdata,*hostout;
		float*devicedata,*deviceout;
		hostdata=(float*)malloc(blockcount*datablocksize);
		hostout=(float*)malloc(blockcount*outblocksize);
		CUDAASSERT(hipMalloc(&devicedata,blockcount*datablocksize));
		CUDAASSERT(hipMalloc(&deviceout,blockcount*outblocksize));
		for (unsigned long long pd=0;pd<nd;pd+=blockcount) {
			int pdo=0;
			for (;pdo<blockcount;pdo++) if (pdo+pd<nd)
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							hostdata[((pc*dy+py)*dx+px)*blockcount+pdo]=data[(((pdo+pd)*dc+pc)*dy+py)*dx+px];
			else break;

			unsigned long long xval=(blockcount*outblocksize/sizeof(float)+CUDALines-1)/CUDALines;
			dim3 gridDim;
			gridDim.y=(xval+1023)/1024; if (xval>=1024) gridDim.x=1024; else gridDim.x=xval;
			gridDim.z=(gridDim.y+1023)/1024; if (gridDim.y>=1024) gridDim.y=1024;
			CUDAASSERT(hipMemcpy(devicedata,hostdata,blockcount*datablocksize,hipMemcpyHostToDevice));

			kmaxblock2D<<<gridDim,CUDALines>>>(devicedata,deviceout,pdo,blockcount,dc,dy,dx,oy,ox,size);
			CUDAASSERT(hipMemcpy(hostout,deviceout,blockcount*outblocksize,hipMemcpyDeviceToHost));

			for (pdo=0;pdo<blockcount;pdo++) if (pdo+pd<nd)
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<oy;py++)
						for (int px=0;px<ox;px++)
							out[(((pdo+pd)*dc+pc)*oy+py)*ox+px]=hostout[((pc*oy+py)*ox+px)*blockcount+pdo];
			else break;
		}
		CUDAASSERT(hipFree(devicedata));
		CUDAASSERT(hipFree(deviceout));
		free(hostdata);
		free(hostout);
		return 0;
	}
	/*
	MaxBlock2D
	This function does a reverse grad passing on max pooling
	*/
	DLL extern int reversemaxblock2D(float*grad,float*data,float*out,int nd,int dc,int dy,int dx,int size)
	{
		int oy=(dy+size-1)/size,ox=(dx+size-1)/size;

		int datablocksize=dc*dy*dx*sizeof(float);
		int outblocksize=dc*oy*ox*sizeof(float);
		int blockcount=CUDAMemory/datablocksize;
		if (blockcount>nd) blockcount=nd+CUDAAligns-1;
		blockcount=blockcount/CUDAAligns*CUDAAligns;
		if (blockcount==0) return INSUFFICIENT_MEMORY;

		float*hostgrad,*hostdata,*hostout;
		float*devicegrad,*devicedata,*deviceout;
		hostgrad=(float*)malloc(blockcount*outblocksize);
		hostdata=(float*)malloc(blockcount*datablocksize);
		hostout=(float*)malloc(blockcount*datablocksize);
		CUDAASSERT(hipMalloc(&devicegrad,blockcount*outblocksize));
		CUDAASSERT(hipMalloc(&devicedata,blockcount*datablocksize));
		CUDAASSERT(hipMalloc(&deviceout,blockcount*datablocksize));

		for (unsigned long long pd=0;pd<nd;pd+=blockcount) {
			int pdo=0;
			for (;pdo<blockcount;pdo++) if (pdo+pd<nd)
			{
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							hostdata[((pc*dy+py)*dx+px)*blockcount+pdo]=data[(((pdo+pd)*dc+pc)*dy+py)*dx+px];
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<oy;py++)
						for (int px=0;px<ox;px++)
							hostgrad[((pc*oy+py)*ox+px)*blockcount+pdo]=grad[(((pdo+pd)*dc+pc)*oy+py)*ox+px];
			}
			else break;

			unsigned long long xval=(blockcount*outblocksize/sizeof(float)+CUDALines-1)/CUDALines;
			dim3 gridDim;
			gridDim.y=(xval+1023)/1024; if (xval>=1024) gridDim.x=1024; else gridDim.x=xval;
			gridDim.z=(gridDim.y+1023)/1024; if (gridDim.y>=1024) gridDim.y=1024;
			CUDAASSERT(hipMemcpy(devicedata,hostdata,blockcount*datablocksize,hipMemcpyHostToDevice));
			CUDAASSERT(hipMemcpy(devicegrad,hostgrad,blockcount*outblocksize,hipMemcpyHostToDevice));

			kreversemaxblock2D<<<gridDim,CUDALines>>>(devicedata,devicegrad,deviceout,NULL,pdo,blockcount,dc,dy,dx,oy,ox,size);
			CUDAASSERT(hipMemcpy(hostout,deviceout,blockcount*datablocksize,hipMemcpyDeviceToHost));

			for (pdo=0;pdo<blockcount;pdo++) if (pdo+pd<nd)
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							out[(((pdo+pd)*dc+pc)*dy+py)*dx+px]=hostout[((pc*dy+py)*dx+px)*blockcount+pdo];
			else break;
		}
		CUDAASSERT(hipFree(devicedata));
		CUDAASSERT(hipFree(deviceout));
		CUDAASSERT(hipFree(devicegrad));
		free(hostdata);
		free(hostout);
		free(hostgrad);
		return 0;
	}
	/*
	This function passes weight or grad by following original max-pooling path
	Likes a inverse version of reversemaxblock2D
	*/
	DLL extern int followmaxblock2D(float*grad,float*data,float*out,int nd,int dc,int dy,int dx,int size)
	{
		int oy=(dy+size-1)/size,ox=(dx+size-1)/size;
		
		int datablocksize=dc*dy*dx*sizeof(float);
		int outblocksize=dc*oy*ox*sizeof(float);
		int blockcount=CUDAMemory/datablocksize;
		if (blockcount>nd) blockcount=nd+CUDAAligns-1;
		blockcount=blockcount/CUDAAligns*CUDAAligns;
		if (blockcount==0) return INSUFFICIENT_MEMORY;

		float*hostgrad,*hostdata,*hostout;
		float*devicegrad,*devicedata,*deviceout;
		hostgrad=(float*)malloc(blockcount*datablocksize);
		hostdata=(float*)malloc(blockcount*datablocksize);
		hostout=(float*)malloc(blockcount*outblocksize);
		CUDAASSERT(hipMalloc(&devicegrad,blockcount*datablocksize));
		CUDAASSERT(hipMalloc(&devicedata,blockcount*datablocksize));
		CUDAASSERT(hipMalloc(&deviceout,blockcount*outblocksize));

		for (unsigned long long pd=0;pd<nd;pd+=blockcount) {
			int pdo=0;
			for (;pdo<blockcount;pdo++) if (pdo+pd<nd)
			{
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							hostdata[((pc*dy+py)*dx+px)*blockcount+pdo]=data[(((pdo+pd)*dc+pc)*dy+py)*dx+px];
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							hostgrad[((pc*dy+py)*dx+px)*blockcount+pdo]=grad[(((pdo+pd)*dc+pc)*dy+py)*dx+px];
			}
			else break;

			unsigned long long xval=(blockcount*outblocksize/sizeof(float)+CUDALines-1)/CUDALines;
			dim3 gridDim;
			gridDim.y=(xval+1023)/1024; if (xval>=1024) gridDim.x=1024; else gridDim.x=xval;
			gridDim.z=(gridDim.y+1023)/1024; if (gridDim.y>=1024) gridDim.y=1024;
			CUDAASSERT(hipMemcpy(devicedata,hostdata,blockcount*datablocksize,hipMemcpyHostToDevice));
			CUDAASSERT(hipMemcpy(devicegrad,hostgrad,blockcount*datablocksize,hipMemcpyHostToDevice));

			kfollowmaxblock2D<<<gridDim,CUDALines>>>(devicedata,devicegrad,deviceout,pdo,blockcount,dc,dy,dx,oy,ox,size);
			CUDAASSERT(hipMemcpy(hostout,deviceout,blockcount*outblocksize,hipMemcpyDeviceToHost));

			for (pdo=0;pdo<blockcount;pdo++) if (pdo+pd<nd)
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<oy;py++)
						for (int px=0;px<ox;px++)
							out[(((pdo+pd)*dc+pc)*oy+py)*ox+px]=hostout[((pc*oy+py)*ox+px)*blockcount+pdo];
			else break;
		}
		CUDAASSERT(hipFree(devicedata));
		CUDAASSERT(hipFree(deviceout));
		CUDAASSERT(hipFree(devicegrad));
		free(hostdata);
		free(hostout);
		free(hostgrad);
		return 0;
	}
	/*
	This function does a square pooling
	*/
	DLL extern int squareblock2D(float*data,float*out,int nd,int dc,int dy,int dx,int size)
	{
		int oy=(dy+size-1)/size,ox=(dx+size-1)/size;

		int datablocksize=dc*dy*dx*sizeof(float);
		int outblocksize=dc*oy*ox*sizeof(float);
		int blockcount=CUDAMemory/datablocksize;
		if (blockcount>nd) blockcount=nd+CUDAAligns-1;
		blockcount=blockcount/CUDAAligns*CUDAAligns;
		if (blockcount==0) return INSUFFICIENT_MEMORY;

		float*hostdata,*hostout;
		float*devicedata,*deviceout;
		hostdata=(float*)malloc(blockcount*datablocksize);
		hostout=(float*)malloc(blockcount*outblocksize);
		CUDAASSERT(hipMalloc(&devicedata,blockcount*datablocksize));
		CUDAASSERT(hipMalloc(&deviceout,blockcount*outblocksize));
		for (unsigned long long pd=0;pd<nd;pd+=blockcount) {
			int pdo=0;
			for (;pdo<blockcount;pdo++) if (pdo+pd<nd)
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							hostdata[((pc*dy+py)*dx+px)*blockcount+pdo]=data[(((pdo+pd)*dc+pc)*dy+py)*dx+px];
			else break;

			unsigned long long xval=(blockcount*outblocksize/sizeof(float)+CUDALines-1)/CUDALines;
			dim3 gridDim;
			gridDim.y=(xval+1023)/1024; if (xval>=1024) gridDim.x=1024; else gridDim.x=xval;
			gridDim.z=(gridDim.y+1023)/1024; if (gridDim.y>=1024) gridDim.y=1024;
			CUDAASSERT(hipMemcpy(devicedata,hostdata,blockcount*datablocksize,hipMemcpyHostToDevice));

			ksquareblock2D<<<gridDim,CUDALines>>>(devicedata,deviceout,pdo,blockcount,dc,dy,dx,oy,ox,size);
			CUDAASSERT(hipMemcpy(hostout,deviceout,blockcount*outblocksize,hipMemcpyDeviceToHost));

			for (pdo=0;pdo<blockcount;pdo++) if (pdo+pd<nd)
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<oy;py++)
						for (int px=0;px<ox;px++)
							out[(((pdo+pd)*dc+pc)*oy+py)*ox+px]=hostout[((pc*oy+py)*ox+px)*blockcount+pdo];
			else break;
		}
		CUDAASSERT(hipFree(devicedata));
		CUDAASSERT(hipFree(deviceout));
		free(hostdata);
		free(hostout);
		return 0;
	}
	/*
	This function does a reverse grad passing on square pooling
	*/
	DLL extern int reversesquareblock2D(float*grad,float*data,float*outdata,float*out,int nd,int dc,int dy,int dx,int size)
	{
		int oy=(dy+size-1)/size,ox=(dx+size-1)/size;

		int datablocksize=dc*dy*dx*sizeof(float);
		int outblocksize=dc*oy*ox*sizeof(float);
		int blockcount=CUDAMemory/datablocksize;
		if (blockcount>nd) blockcount=nd+CUDAAligns-1;
		blockcount=blockcount/CUDAAligns*CUDAAligns;
		if (blockcount==0) return INSUFFICIENT_MEMORY;

		float*hostgrad,*hostdata,*hostoutdata,*hostout;
		float*devicegrad,*devicedata,*deviceoutdata,*deviceout;
		hostgrad=(float*)malloc(blockcount*outblocksize);
		hostdata=(float*)malloc(blockcount*datablocksize);
		hostout=(float*)malloc(blockcount*datablocksize);
		hostoutdata=(float*)malloc(blockcount*outblocksize);
		CUDAASSERT(hipMalloc(&devicegrad,blockcount*outblocksize));
		CUDAASSERT(hipMalloc(&devicedata,blockcount*datablocksize));
		CUDAASSERT(hipMalloc(&deviceout,blockcount*datablocksize));
		CUDAASSERT(hipMalloc(&deviceoutdata,blockcount*outblocksize));

		for (unsigned long long pd=0;pd<nd;pd+=blockcount) {
			int pdo=0;
			for (;pdo<blockcount;pdo++) if (pdo+pd<nd)
			{
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							hostdata[((pc*dy+py)*dx+px)*blockcount+pdo]=data[(((pdo+pd)*dc+pc)*dy+py)*dx+px];
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<oy;py++)
						for (int px=0;px<ox;px++)
							hostgrad[((pc*oy+py)*ox+px)*blockcount+pdo]=grad[(((pdo+pd)*dc+pc)*oy+py)*ox+px];
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<oy;py++)
						for (int px=0;px<ox;px++)
							hostoutdata[((pc*oy+py)*ox+px)*blockcount+pdo]=outdata[(((pdo+pd)*dc+pc)*oy+py)*ox+px];
			}
			else break;

			unsigned long long xval=(blockcount*outblocksize/sizeof(float)+CUDALines-1)/CUDALines;
			dim3 gridDim;
			gridDim.y=(xval+1023)/1024; if (xval>=1024) gridDim.x=1024; else gridDim.x=xval;
			gridDim.z=(gridDim.y+1023)/1024; if (gridDim.y>=1024) gridDim.y=1024;
			CUDAASSERT(hipMemcpy(devicedata,hostdata,blockcount*datablocksize,hipMemcpyHostToDevice));
			CUDAASSERT(hipMemcpy(devicegrad,hostgrad,blockcount*outblocksize,hipMemcpyHostToDevice));
			CUDAASSERT(hipMemcpy(deviceoutdata,hostoutdata,blockcount*outblocksize,hipMemcpyHostToDevice));

			kreversesquareblock2D<<<gridDim,CUDALines>>>(devicedata,devicegrad,deviceout,deviceoutdata,pdo,blockcount,dc,dy,dx,oy,ox,size);
			CUDAASSERT(hipMemcpy(hostout,deviceout,blockcount*datablocksize,hipMemcpyDeviceToHost));

			for (pdo=0;pdo<blockcount;pdo++) if (pdo+pd<nd)
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							out[(((pdo+pd)*dc+pc)*dy+py)*dx+px]=hostout[((pc*dy+py)*dx+px)*blockcount+pdo];
			else break;
		}
		CUDAASSERT(hipFree(devicedata));
		CUDAASSERT(hipFree(deviceout));
		CUDAASSERT(hipFree(devicegrad));
		CUDAASSERT(hipFree(deviceoutdata));
		free(hostdata);
		free(hostout);
		free(hostgrad);
		free(hostoutdata);
		return 0;
	}
	/*
	This function passes weight or grad by following original square-pooling path
	Likes a inverse version of reversesquareblock2D
	*/
	DLL extern int followsquareblock2D(float*grad,float*data,float*outdata,float*out,int nd,int dc,int dy,int dx,int size)
	{
		int oy=(dy+size-1)/size,ox=(dx+size-1)/size;

		int datablocksize=dc*dy*dx*sizeof(float);
		int outblocksize=dc*oy*ox*sizeof(float);
		int blockcount=CUDAMemory/datablocksize;
		if (blockcount>nd) blockcount=nd+CUDAAligns-1;
		blockcount=blockcount/CUDAAligns*CUDAAligns;
		if (blockcount==0) return INSUFFICIENT_MEMORY;

		float*hostgrad,*hostdata,*hostoutdata,*hostout;
		float*devicegrad,*devicedata,*deviceoutdata,*deviceout;
		hostgrad=(float*)malloc(blockcount*datablocksize);
		hostdata=(float*)malloc(blockcount*datablocksize);
		hostout=(float*)malloc(blockcount*outblocksize);
		hostoutdata=(float*)malloc(blockcount*outblocksize);
		CUDAASSERT(hipMalloc(&devicegrad,blockcount*datablocksize));
		CUDAASSERT(hipMalloc(&devicedata,blockcount*datablocksize));
		CUDAASSERT(hipMalloc(&deviceout,blockcount*outblocksize));
		CUDAASSERT(hipMalloc(&deviceoutdata,blockcount*outblocksize));

		for (unsigned long long pd=0;pd<nd;pd+=blockcount) {
			int pdo=0;
			for (;pdo<blockcount;pdo++) if (pdo+pd<nd)
			{
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							hostdata[((pc*dy+py)*dx+px)*blockcount+pdo]=data[(((pdo+pd)*dc+pc)*dy+py)*dx+px];
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							hostgrad[((pc*dy+py)*dx+px)*blockcount+pdo]=grad[(((pdo+pd)*dc+pc)*dy+py)*dx+px];
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<oy;py++)
						for (int px=0;px<ox;px++)
							hostoutdata[((pc*oy+py)*ox+px)*blockcount+pdo]=outdata[(((pdo+pd)*dc+pc)*oy+py)*ox+px];
			}
			else break;

			unsigned long long xval=(blockcount*outblocksize/sizeof(float)+CUDALines-1)/CUDALines;
			dim3 gridDim;
			gridDim.y=(xval+1023)/1024; if (xval>=1024) gridDim.x=1024; else gridDim.x=xval;
			gridDim.z=(gridDim.y+1023)/1024; if (gridDim.y>=1024) gridDim.y=1024;
			CUDAASSERT(hipMemcpy(devicedata,hostdata,blockcount*datablocksize,hipMemcpyHostToDevice));
			CUDAASSERT(hipMemcpy(devicegrad,hostgrad,blockcount*datablocksize,hipMemcpyHostToDevice));
			CUDAASSERT(hipMemcpy(deviceoutdata,hostoutdata,blockcount*outblocksize,hipMemcpyHostToDevice));

			kfollowsquareblock2D<<<gridDim,CUDALines>>>(devicedata,devicegrad,deviceout,deviceoutdata,pdo,blockcount,dc,dy,dx,oy,ox,size);
			CUDAASSERT(hipMemcpy(hostout,deviceout,blockcount*outblocksize,hipMemcpyDeviceToHost));

			for (pdo=0;pdo<blockcount;pdo++) if (pdo+pd<nd)
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<oy;py++)
						for (int px=0;px<ox;px++)
							out[(((pdo+pd)*dc+pc)*oy+py)*ox+px]=hostout[((pc*oy+py)*ox+px)*blockcount+pdo];
			else break;
		}
		CUDAASSERT(hipFree(devicedata));
		CUDAASSERT(hipFree(deviceout));
		CUDAASSERT(hipFree(devicegrad));
		CUDAASSERT(hipFree(deviceoutdata));
		free(hostdata);
		free(hostout);
		free(hostgrad);
		free(hostoutdata);
		return 0;
	}
	/*
	This function does a cross-layer square pooling
	*/
	DLL extern int squarelayer(float*data,float*out,int nd,int dc,int dy,int dx,int size)
	{
		int oc=dc/size;

		int datablocksize=dc*dy*dx*sizeof(float);
		int outblocksize=oc*dy*dx*sizeof(float);
		int blockcount=CUDAMemory/datablocksize;
		if (blockcount>nd) blockcount=nd+CUDAAligns-1;
		blockcount=blockcount/CUDAAligns*CUDAAligns;
		if (blockcount==0) return INSUFFICIENT_MEMORY;

		float*hostdata,*hostout;
		float*devicedata,*deviceout;
		hostdata=(float*)malloc(blockcount*datablocksize);
		hostout=(float*)malloc(blockcount*outblocksize);
		CUDAASSERT(hipMalloc(&devicedata,blockcount*datablocksize));
		CUDAASSERT(hipMalloc(&deviceout,blockcount*outblocksize));
		for (unsigned long long pd=0;pd<nd;pd+=blockcount) {
			int pdo=0;
			for (;pdo<blockcount;pdo++) if (pdo+pd<nd)
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							hostdata[((pc*dy+py)*dx+px)*blockcount+pdo]=data[(((pdo+pd)*dc+pc)*dy+py)*dx+px];
			else break;

			unsigned long long xval=(blockcount*outblocksize/sizeof(float)+CUDALines-1)/CUDALines;
			dim3 gridDim;
			gridDim.y=(xval+1023)/1024; if (xval>=1024) gridDim.x=1024; else gridDim.x=xval;
			gridDim.z=(gridDim.y+1023)/1024; if (gridDim.y>=1024) gridDim.y=1024;
			CUDAASSERT(hipMemcpy(devicedata,hostdata,blockcount*datablocksize,hipMemcpyHostToDevice));

			ksquarelayer<<<gridDim,CUDALines>>>(devicedata,deviceout,pdo,blockcount,oc,dy*dx,size);
			CUDAASSERT(hipMemcpy(hostout,deviceout,blockcount*outblocksize,hipMemcpyDeviceToHost));

			for (pdo=0;pdo<blockcount;pdo++) if (pdo+pd<nd)
				for (int pc=0;pc<oc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							out[(((pdo+pd)*oc+pc)*dy+py)*dx+px]=hostout[((pc*dy+py)*dx+px)*blockcount+pdo];
			else break;
		}
		CUDAASSERT(hipFree(devicedata));
		CUDAASSERT(hipFree(deviceout));
		free(hostdata);
		free(hostout);
		return 0;
	}
	/*
	This function does a grad passing on a cross-layer square pooling
	*/
	DLL extern int reversesquarelayer(float*grad,float*data,float*outdata,float*out,int nd,int dc,int dy,int dx,int size)
	{
		int oc=dc/size;

		int datablocksize=dc*dy*dx*sizeof(float);
		int outblocksize=oc*dy*dx*sizeof(float);
		int blockcount=CUDAMemory/datablocksize;
		if (blockcount>nd) blockcount=nd+CUDAAligns-1;
		blockcount=blockcount/CUDAAligns*CUDAAligns;
		if (blockcount==0) return INSUFFICIENT_MEMORY;

		float*hostgrad,*hostdata,*hostoutdata,*hostout;
		float*devicegrad,*devicedata,*deviceoutdata,*deviceout;
		hostgrad=(float*)malloc(blockcount*outblocksize);
		hostdata=(float*)malloc(blockcount*datablocksize);
		hostout=(float*)malloc(blockcount*datablocksize);
		hostoutdata=(float*)malloc(blockcount*outblocksize);
		CUDAASSERT(hipMalloc(&devicegrad,blockcount*outblocksize));
		CUDAASSERT(hipMalloc(&devicedata,blockcount*datablocksize));
		CUDAASSERT(hipMalloc(&deviceout,blockcount*datablocksize));
		CUDAASSERT(hipMalloc(&deviceoutdata,blockcount*outblocksize));

		for (unsigned long long pd=0;pd<nd;pd+=blockcount) {
			int pdo=0;
			for (;pdo<blockcount;pdo++) if (pdo+pd<nd)
			{
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							hostdata[((pc*dy+py)*dx+px)*blockcount+pdo]=data[(((pdo+pd)*dc+pc)*dy+py)*dx+px];
				for (int pc=0;pc<oc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							hostgrad[((pc*dy+py)*dx+px)*blockcount+pdo]=grad[(((pdo+pd)*oc+pc)*dy+py)*dx+px];
				for (int pc=0;pc<oc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							hostoutdata[((pc*dy+py)*dx+px)*blockcount+pdo]=outdata[(((pdo+pd)*oc+pc)*dy+py)*dx+px];
			}
			else break;

			unsigned long long xval=(blockcount*outblocksize/sizeof(float)+CUDALines-1)/CUDALines;
			dim3 gridDim;
			gridDim.y=(xval+1023)/1024; if (xval>=1024) gridDim.x=1024; else gridDim.x=xval;
			gridDim.z=(gridDim.y+1023)/1024; if (gridDim.y>=1024) gridDim.y=1024;
			CUDAASSERT(hipMemcpy(devicedata,hostdata,blockcount*datablocksize,hipMemcpyHostToDevice));
			CUDAASSERT(hipMemcpy(devicegrad,hostgrad,blockcount*outblocksize,hipMemcpyHostToDevice));
			CUDAASSERT(hipMemcpy(deviceoutdata,hostoutdata,blockcount*outblocksize,hipMemcpyHostToDevice));

			kreversesquarelayer<<<gridDim,CUDALines>>>(devicedata,devicegrad,deviceout,deviceoutdata,pdo,blockcount,oc,dy*dx,size);
			CUDAASSERT(hipMemcpy(hostout,deviceout,blockcount*datablocksize,hipMemcpyDeviceToHost));

			for (pdo=0;pdo<blockcount;pdo++) if (pdo+pd<nd)
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							out[(((pdo+pd)*dc+pc)*dy+py)*dx+px]=hostout[((pc*dy+py)*dx+px)*blockcount+pdo];
			else break;
		}
		CUDAASSERT(hipFree(devicedata));
		CUDAASSERT(hipFree(deviceout));
		CUDAASSERT(hipFree(devicegrad));
		CUDAASSERT(hipFree(deviceoutdata));
		free(hostdata);
		free(hostout);
		free(hostgrad);
		free(hostoutdata);
		return 0;
	}
	/*
	This function passes weight or grad by following original cross-layer square pooling path
	Likes a inverse version of reversesquarelayer
	*/
	DLL extern int followsquarelayer(float*grad,float*data,float*outdata,float*out,int nd,int dc,int dy,int dx,int size)
	{
		int oc=dc/size;

		int datablocksize=dc*dy*dx*sizeof(float);
		int outblocksize=oc*dy*dx*sizeof(float);
		int blockcount=CUDAMemory/datablocksize;
		if (blockcount>nd) blockcount=nd+CUDAAligns-1;
		blockcount=blockcount/CUDAAligns*CUDAAligns;
		if (blockcount==0) return INSUFFICIENT_MEMORY;

		float*hostgrad,*hostdata,*hostoutdata,*hostout;
		float*devicegrad,*devicedata,*deviceoutdata,*deviceout;
		hostgrad=(float*)malloc(blockcount*datablocksize);
		hostdata=(float*)malloc(blockcount*datablocksize);
		hostout=(float*)malloc(blockcount*outblocksize);
		hostoutdata=(float*)malloc(blockcount*outblocksize);
		CUDAASSERT(hipMalloc(&devicegrad,blockcount*datablocksize));
		CUDAASSERT(hipMalloc(&devicedata,blockcount*datablocksize));
		CUDAASSERT(hipMalloc(&deviceout,blockcount*outblocksize));
		CUDAASSERT(hipMalloc(&deviceoutdata,blockcount*outblocksize));

		for (unsigned long long pd=0;pd<nd;pd+=blockcount) {
			int pdo=0;
			for (;pdo<blockcount;pdo++) if (pdo+pd<nd)
			{
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							hostdata[((pc*dy+py)*dx+px)*blockcount+pdo]=data[(((pdo+pd)*dc+pc)*dy+py)*dx+px];
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							hostgrad[((pc*dy+py)*dx+px)*blockcount+pdo]=grad[(((pdo+pd)*dc+pc)*dy+py)*dx+px];
				for (int pc=0;pc<oc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							hostoutdata[((pc*dy+py)*dx+px)*blockcount+pdo]=outdata[(((pdo+pd)*oc+pc)*dy+py)*dx+px];
			}
			else break;

			unsigned long long xval=(blockcount*outblocksize/sizeof(float)+CUDALines-1)/CUDALines;
			dim3 gridDim;
			gridDim.y=(xval+1023)/1024; if (xval>=1024) gridDim.x=1024; else gridDim.x=xval;
			gridDim.z=(gridDim.y+1023)/1024; if (gridDim.y>=1024) gridDim.y=1024;
			CUDAASSERT(hipMemcpy(devicedata,hostdata,blockcount*datablocksize,hipMemcpyHostToDevice));
			CUDAASSERT(hipMemcpy(devicegrad,hostgrad,blockcount*datablocksize,hipMemcpyHostToDevice));
			CUDAASSERT(hipMemcpy(deviceoutdata,hostoutdata,blockcount*outblocksize,hipMemcpyHostToDevice));

			kfollowsquarelayer<<<gridDim,CUDALines>>>(devicedata,devicegrad,deviceout,deviceoutdata,pdo,blockcount,oc,dy*dx,size);
			CUDAASSERT(hipMemcpy(hostout,deviceout,blockcount*outblocksize,hipMemcpyDeviceToHost));

			for (pdo=0;pdo<blockcount;pdo++) if (pdo+pd<nd)
				for (int pc=0;pc<oc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							out[(((pdo+pd)*oc+pc)*dy+py)*dx+px]=hostout[((pc*dy+py)*dx+px)*blockcount+pdo];
			else break;
		}
		CUDAASSERT(hipFree(devicedata));
		CUDAASSERT(hipFree(deviceout));
		CUDAASSERT(hipFree(devicegrad));
		CUDAASSERT(hipFree(deviceoutdata));
		free(hostdata);
		free(hostout);
		free(hostgrad);
		free(hostoutdata);
		return 0;
	}
	/*
	This function does a cross-layer max pooling
	*/
	DLL extern int maxlayer(float*data,float*out,int nd,int dc,int dy,int dx,int size)
	{
		int oc=dc/size;

		int datablocksize=dc*dy*dx*sizeof(float);
		int outblocksize=oc*dy*dx*sizeof(float);
		int blockcount=CUDAMemory/datablocksize;
		if (blockcount>nd) blockcount=nd+CUDAAligns-1;
		blockcount=blockcount/CUDAAligns*CUDAAligns;
		if (blockcount==0) return INSUFFICIENT_MEMORY;

		float*hostdata,*hostout;
		float*devicedata,*deviceout;
		hostdata=(float*)malloc(blockcount*datablocksize);
		hostout=(float*)malloc(blockcount*outblocksize);
		CUDAASSERT(hipMalloc(&devicedata,blockcount*datablocksize));
		CUDAASSERT(hipMalloc(&deviceout,blockcount*outblocksize));
		for (unsigned long long pd=0;pd<nd;pd+=blockcount) {
			int pdo=0;
			for (;pdo<blockcount;pdo++) if (pdo+pd<nd)
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							hostdata[((pc*dy+py)*dx+px)*blockcount+pdo]=data[(((pdo+pd)*dc+pc)*dy+py)*dx+px];
			else break;

			unsigned long long xval=(blockcount*outblocksize/sizeof(float)+CUDALines-1)/CUDALines;
			dim3 gridDim;
			gridDim.y=(xval+1023)/1024; if (xval>=1024) gridDim.x=1024; else gridDim.x=xval;
			gridDim.z=(gridDim.y+1023)/1024; if (gridDim.y>=1024) gridDim.y=1024;
			CUDAASSERT(hipMemcpy(devicedata,hostdata,blockcount*datablocksize,hipMemcpyHostToDevice));

			kmaxlayer<<<gridDim,CUDALines>>>(devicedata,deviceout,pdo,blockcount,oc,dy*dx,size);
			CUDAASSERT(hipMemcpy(hostout,deviceout,blockcount*outblocksize,hipMemcpyDeviceToHost));

			for (pdo=0;pdo<blockcount;pdo++) if (pdo+pd<nd)
				for (int pc=0;pc<oc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							out[(((pdo+pd)*oc+pc)*dy+py)*dx+px]=hostout[((pc*dy+py)*dx+px)*blockcount+pdo];
			else break;
		}
		CUDAASSERT(hipFree(devicedata));
		CUDAASSERT(hipFree(deviceout));
		free(hostdata);
		free(hostout);
		return 0;
	}
	/*
	This function does a grad passing on a cross-layer max pooling
	*/
	DLL extern int reversemaxlayer(float*grad,float*data,float*out,int nd,int dc,int dy,int dx,int size)
	{
		int oc=dc/size;

		int datablocksize=dc*dy*dx*sizeof(float);
		int outblocksize=oc*dy*dx*sizeof(float);
		int blockcount=CUDAMemory/datablocksize;
		if (blockcount>nd) blockcount=nd+CUDAAligns-1;
		blockcount=blockcount/CUDAAligns*CUDAAligns;
		if (blockcount==0) return INSUFFICIENT_MEMORY;

		float*hostgrad,*hostdata,*hostout;
		float*devicegrad,*devicedata,*deviceout;
		hostgrad=(float*)malloc(blockcount*outblocksize);
		hostdata=(float*)malloc(blockcount*datablocksize);
		hostout=(float*)malloc(blockcount*datablocksize);
		CUDAASSERT(hipMalloc(&devicegrad,blockcount*outblocksize));
		CUDAASSERT(hipMalloc(&devicedata,blockcount*datablocksize));
		CUDAASSERT(hipMalloc(&deviceout,blockcount*datablocksize));

		for (unsigned long long pd=0;pd<nd;pd+=blockcount) {
			int pdo=0;
			for (;pdo<blockcount;pdo++) if (pdo+pd<nd)
			{
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							hostdata[((pc*dy+py)*dx+px)*blockcount+pdo]=data[(((pdo+pd)*dc+pc)*dy+py)*dx+px];
				for (int pc=0;pc<oc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							hostgrad[((pc*dy+py)*dx+px)*blockcount+pdo]=grad[(((pdo+pd)*oc+pc)*dy+py)*dx+px];
			}
			else break;

			unsigned long long xval=(blockcount*outblocksize/sizeof(float)+CUDALines-1)/CUDALines;
			dim3 gridDim;
			gridDim.y=(xval+1023)/1024; if (xval>=1024) gridDim.x=1024; else gridDim.x=xval;
			gridDim.z=(gridDim.y+1023)/1024; if (gridDim.y>=1024) gridDim.y=1024;
			CUDAASSERT(hipMemcpy(devicedata,hostdata,blockcount*datablocksize,hipMemcpyHostToDevice));
			CUDAASSERT(hipMemcpy(devicegrad,hostgrad,blockcount*outblocksize,hipMemcpyHostToDevice));

			kreversemaxlayer<<<gridDim,CUDALines>>>(devicedata,devicegrad,deviceout,pdo,blockcount,oc,dy*dx,size);
			CUDAASSERT(hipMemcpy(hostout,deviceout,blockcount*datablocksize,hipMemcpyDeviceToHost));

			for (pdo=0;pdo<blockcount;pdo++) if (pdo+pd<nd)
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							out[(((pdo+pd)*dc+pc)*dy+py)*dx+px]=hostout[((pc*dy+py)*dx+px)*blockcount+pdo];
			else break;
		}
		CUDAASSERT(hipFree(devicedata));
		CUDAASSERT(hipFree(deviceout));
		CUDAASSERT(hipFree(devicegrad));
		free(hostdata);
		free(hostout);
		free(hostgrad);
		return 0;
	}
	/*
	This function passes weight or grad by following original cross-layer max pooling path
	Likes a inverse version of reversemaxlayer
	*/
	DLL extern int followmaxlayer(float*grad,float*data,float*out,int nd,int dc,int dy,int dx,int size)
	{
		int oc=dc/size;

		int datablocksize=dc*dy*dx*sizeof(float);
		int outblocksize=oc*dy*dx*sizeof(float);
		int blockcount=CUDAMemory/datablocksize;
		if (blockcount>nd) blockcount=nd+CUDAAligns-1;
		blockcount=blockcount/CUDAAligns*CUDAAligns;
		if (blockcount==0) return INSUFFICIENT_MEMORY;

		float*hostgrad,*hostdata,*hostout;
		float*devicegrad,*devicedata,*deviceout;
		hostgrad=(float*)malloc(blockcount*datablocksize);
		hostdata=(float*)malloc(blockcount*datablocksize);
		hostout=(float*)malloc(blockcount*outblocksize);
		CUDAASSERT(hipMalloc(&devicegrad,blockcount*datablocksize));
		CUDAASSERT(hipMalloc(&devicedata,blockcount*datablocksize));
		CUDAASSERT(hipMalloc(&deviceout,blockcount*outblocksize));

		for (unsigned long long pd=0;pd<nd;pd+=blockcount) {
			int pdo=0;
			for (;pdo<blockcount;pdo++) if (pdo+pd<nd)
			{
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							hostdata[((pc*dy+py)*dx+px)*blockcount+pdo]=data[(((pdo+pd)*dc+pc)*dy+py)*dx+px];
				for (int pc=0;pc<dc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							hostgrad[((pc*dy+py)*dx+px)*blockcount+pdo]=grad[(((pdo+pd)*dc+pc)*dy+py)*dx+px];
			}
			else break;

			unsigned long long xval=(blockcount*outblocksize/sizeof(float)+CUDALines-1)/CUDALines;
			dim3 gridDim;
			gridDim.y=(xval+1023)/1024; if (xval>=1024) gridDim.x=1024; else gridDim.x=xval;
			gridDim.z=(gridDim.y+1023)/1024; if (gridDim.y>=1024) gridDim.y=1024;
			CUDAASSERT(hipMemcpy(devicedata,hostdata,blockcount*datablocksize,hipMemcpyHostToDevice));
			CUDAASSERT(hipMemcpy(devicegrad,hostgrad,blockcount*datablocksize,hipMemcpyHostToDevice));

			kfollowmaxlayer<<<gridDim,CUDALines>>>(devicedata,devicegrad,deviceout,pdo,blockcount,oc,dy*dx,size);
			CUDAASSERT(hipMemcpy(hostout,deviceout,blockcount*outblocksize,hipMemcpyDeviceToHost));

			for (pdo=0;pdo<blockcount;pdo++) if (pdo+pd<nd)
				for (int pc=0;pc<oc;pc++)
					for (int py=0;py<dy;py++)
						for (int px=0;px<dx;px++)
							out[(((pdo+pd)*oc+pc)*dy+py)*dx+px]=hostout[((pc*dy+py)*dx+px)*blockcount+pdo];
			else break;
		}
		CUDAASSERT(hipFree(devicedata));
		CUDAASSERT(hipFree(deviceout));
		CUDAASSERT(hipFree(devicegrad));
		free(hostdata);
		free(hostout);
		free(hostgrad);
		return 0;
	}
	DLL extern int removeDC(float*data,float*out,int nd,int count)
	{
		int blockcount=CUDAMemory/(count*sizeof(float));
		if (blockcount==0) return INSUFFICIENT_MEMORY;

		float*devicedata,*devicecount;

		CUDAASSERT(hipMalloc(&devicedata,blockcount*count*sizeof(float)));
		CUDAASSERT(hipMalloc(&devicecount,count*sizeof(float)));
		CUDAASSERT(hipMemset(devicecount,0,count*sizeof(float)));
		unsigned long long xval=(count+CUDALines-1)/CUDALines;
		dim3 gridDim;
		gridDim.y=(xval+1023)/1024; if (xval>=1024) gridDim.x=1024; else gridDim.x=xval;
		gridDim.z=(gridDim.y+1023)/1024; if (gridDim.y>=1024) gridDim.y=1024;

		for (unsigned long long pd=0;pd<nd;pd+=blockcount)
		{
			int pdo=min(blockcount,(int)(nd-pd));
			CUDAASSERT(hipMemcpy(devicedata,data+pd*count,pdo*count*sizeof(float),hipMemcpyHostToDevice));
			kaccumDC<<<gridDim,CUDALines>>>(devicedata,devicecount,pdo,count);
		}
		CUDAASSERT(hipDeviceSynchronize());
		kdivi<<<gridDim,CUDALines>>>(devicecount,nd,count);
		CUDAASSERT(hipDeviceSynchronize());
		for (unsigned long long pd=0;pd<nd;pd+=blockcount)
		{
			int pdo=min(blockcount,(int)(nd-pd));
			CUDAASSERT(hipMemcpy(devicedata,data+pd*count,pdo*count*sizeof(float),hipMemcpyHostToDevice));
			ksub<<<gridDim,CUDALines>>>(devicedata,devicecount,pdo,count);
			CUDAASSERT(hipMemcpy(out+pd*count,devicedata,pdo*count*sizeof(float),hipMemcpyDeviceToHost));
		}
		CUDAASSERT(hipFree(devicedata));
		CUDAASSERT(hipFree(devicecount));

		return 0;
	}
	/*
	Block permutation (on cpu), order is what data i will be
	*/
	DLL extern int blockpermutation(float*data,int*order,int len,int count)
	{
		int*flags,*flagorder;
		flags=(int*)malloc(len*sizeof(int));
		flagorder=(int*)malloc(len*sizeof(int));
		memset(flags,0,len*sizeof(int));
		for (int i=0;i<len;i++)
		{
			if (order[i]>=len||order[i]<0) {
				free(flags);
				free(flagorder);
				return INVALID_ARGUMENT;
			}
			flags[order[i]]++;
			flagorder[order[i]]=i;
		}

		for (int i=0;i<len;i++)
			if (flags[i]!=1) {
				free(flags);
				free(flagorder);
				return INVALID_ARGUMENT;
			}

		memset(flags,0,len*sizeof(int));

		float*tmp;
		tmp=(float*)malloc(count*sizeof(float));

		for (int i=0;i<len;i++)
		{
			if (flags[i]==1) continue;
			if (flagorder[i]==i) continue;
			memcpy(tmp,&data[((unsigned long long )(i))*count],count*sizeof(float));
			int p;
			for (p=i;flags[flagorder[p]]==0;p=flagorder[p])
			{
				flags[p]=1;
				memcpy(&data[((unsigned long long )(p))*count],&data[((unsigned long long )(flagorder[p]))*count],count*sizeof(float));
			}
			memcpy(&data[((unsigned long long )(p))*count],tmp,count*sizeof(float));
		}

		free(flags);
		free(flagorder);
		free(tmp);

		return 0;
	}
	DLL extern int normalize(float* weight, int out, int in)
	{
		float* datapool;
		int blocksize=65536;
		if (blocksize*in*sizeof(float)>CUDAMemory)
			blocksize = CUDAMemory/in/sizeof(float);
		if (blocksize==0) return INSUFFICIENT_MEMORY;

		CUDAASSERT(hipMalloc(&datapool,blocksize*in*sizeof(float)));
		for (unsigned long long pd=0;pd<out;pd+=blocksize) {
			CUDAASSERT(hipMemcpy(datapool,weight+pd*in,(pd+blocksize>=out?out-pd:blocksize)*in*sizeof(float),hipMemcpyHostToDevice));
			normalize_kern<<<((pd+blocksize>=out?out-pd:blocksize)+CUDALines-1)/CUDALines,CUDALines>>>(datapool,pd+blocksize>=out?out-pd:blocksize,in);
			CUDAASSERT(hipMemcpy(weight+pd*in,datapool,(pd+blocksize>=out?out-pd:blocksize)*in*sizeof(float),hipMemcpyDeviceToHost));
		}
		CUDAASSERT(hipFree(datapool));

		return 0;
	}
	/*CPU Function: balance weights*/
	DLL extern void balance(float* weight, int count,int dim)
	{
		for (unsigned long long i=0;i<count;i++)
		{
			float sum=0;
			for (int j=0;j<dim;j++) sum+=weight[i*dim+j];
			sum/=dim;
			for (int j=0;j<dim;j++) weight[i*dim+j]-=sum;
		}
	}
	/*
	This function does a general map-style transformation
	op:
	0:<1
	1:<[d0]
	2:<[d1]
	3:<[d2]
	4:<[d3]
	5:<[(float)next 4 byte]
	6:<id
	7:<<+>
	8:<<->
	9:<->
	10:<<*>
	11:<</>
	12:<<%>
	13:<sin>
	14:<cos>
	15:<tan>
	16:<cot>
	17:<sec>
	18:<csc>
	19:<1/(.+eps)>
	20:<1+>
	21:<exp>
	22:<log>
	23:<sinh>
	24:<cosh>
	25:<tanh>
	26:<coth>
	27:<sech>
	28:<csch>
	29:<sqr>
	30:<sqrt>
	31:<<pow>
	32:<asin>
	33:<acos>
	34:<atan>
	35:<fabs>
	36:<ceil>
	37:<floor>
	38:<sigmoid>
	39:<x(1-x)>
	40:<dup-1>>
	41:<dup-2>>
	42:<dup-3>>
	43:<dup-4>>
	44:<<(>)>   (not float out)
	45:<<(<)>   (not float out)
	46:<<(== (1e-6))>   (not float out)
	47:<<(>= (1e-6))>   (not float out)
	48:<<(<= (1e-6))>   (not float out)
	49:<<(!= (1e-6))>   (not float out)
	50:<<(== (exact)>	(not float out)
	51:<<and>   (not float out)
	52:<<or>   (not float out)
	53:<<xor>   (not float out)
	54:<not>   (not float out)
	55:<<<[0]?[-1]:[-2]>
	56:<<swap>>
	57:<isnan>   (not float out)
	58:<isinf>    (not float out)

	Every function with multiple args, it is feed on sequence of poping stack
	returns on stack value
	*/
	DLL extern int transform(float*d0,unsigned char*operates,int oplen,unsigned long size,float*out,float*d1,float*d2,float*d3)
	{
		if (oplen>MAX_TRANSFORM) return INVALID_ARGUMENT;
		__set_op_transform(operates,oplen);
		//Scan how many variables needed
		float*darr[4]={d0,d1,d2,d3};
		int largest=0;
		for (int p=0;p<oplen;p++)
			if (operates[p]==5) p+=4;
			else if (operates[p]<5)
				if (operates[p]>largest) largest=operates[p];
		float*deviced[4];
		for (int i=0;i<largest;i++)
			CUDAASSERT(hipMalloc(&deviced[i],CUDAMemory));
		float*deviceout;
		CUDAASSERT(hipMalloc(&deviceout,CUDAMemory));
		int sect=CUDAMemory/sizeof(float)/(largest==0?1:largest);
		if (sect>1024*256*CUDAAligns) sect=1024*256*CUDAAligns;
		int dsect=(sect+CUDALines-1)/CUDALines;
		for (unsigned long long ps=0;ps<size;ps+=sect)
		{
			for (int i=0;i<largest;i++)
				CUDAASSERT(hipMemcpy(deviced[i],&darr[i][ps],((size-ps>=sect)?sect:(size-ps))*sizeof(float),hipMemcpyHostToDevice));
			dim3 gridDim;

			gridDim.y=(dsect+1023)/1024; if (dsect>=1024) gridDim.x=1024; else gridDim.x=dsect;
			gridDim.z=(gridDim.y+1023)/1024; if (gridDim.y>=1024) gridDim.y=1024;
			sprintf(custom,"(%d,%d,%d,%d)",gridDim.x,gridDim.y,gridDim.z,CUDAAligns);
			ktransform<<<gridDim,CUDALines>>>(deviced[0],deviced[1],deviced[2],deviced[3],deviceout,(size-ps>=sect)?sect:(size-ps),oplen,ps);
			CUDAASSERT(hipMemcpy(&out[ps],deviceout,((size-ps>=sect)?sect:(size-ps))*sizeof(float),hipMemcpyDeviceToHost));
		}
		for (int i=0;i<largest;i++)
			CUDAASSERT(hipFree(deviced[i]));
		CUDAASSERT(hipFree(deviceout));
		return 0;
	}
	DLL extern int transformD(double*d0,unsigned char*operates,int oplen,unsigned long size,double*out,double*d1,double*d2,double*d3)
	{
		if (oplen>MAX_TRANSFORM) return INVALID_ARGUMENT;
		__set_op_transform(operates,oplen);
		//Scan how many variables needed
		double*darr[4]={d0,d1,d2,d3};
		int largest=0;
		for (int p=0;p<oplen;p++)
			if (operates[p]==5) p+=8;
			else if (operates[p]<5)
				if (operates[p]>largest) largest=operates[p];
		double*deviced[4];
		for (int i=0;i<largest;i++)
			CUDAASSERT(hipMalloc(&deviced[i],CUDAMemory));
		double*deviceout;
		CUDAASSERT(hipMalloc(&deviceout,CUDAMemory));
		int sect=CUDAMemory/sizeof(double)/(largest==0?1:largest);
		if (sect>1024*256*CUDAAligns) sect=1024*256*CUDAAligns;
		int dsect=(sect+CUDALines-1)/CUDALines;
		for (unsigned long long ps=0;ps<size;ps+=sect)
		{
			for (int i=0;i<largest;i++)
				CUDAASSERT(hipMemcpy(deviced[i],&darr[i][ps],((size-ps>=sect)?sect:(size-ps))*sizeof(double),hipMemcpyHostToDevice));
			dim3 gridDim;

			gridDim.y=(dsect+1023)/1024; if (dsect>=1024) gridDim.x=1024; else gridDim.x=dsect;
			gridDim.z=(gridDim.y+1023)/1024; if (gridDim.y>=1024) gridDim.y=1024;
			sprintf(custom,"(%d,%d,%d,%d)",gridDim.x,gridDim.y,gridDim.z,CUDAAligns);
			ktransformD<<<gridDim,CUDALines>>>(deviced[0],deviced[1],deviced[2],deviced[3],deviceout,(size-ps>=sect)?sect:(size-ps),oplen,ps);
			CUDAASSERT(hipMemcpy(&out[ps],deviceout,((size-ps>=sect)?sect:(size-ps))*sizeof(double),hipMemcpyDeviceToHost));
		}
		for (int i=0;i<largest;i++)
			CUDAASSERT(hipFree(deviced[i]));
		CUDAASSERT(hipFree(deviceout));
		return 0;
	}
    DLL extern int transformgpu(float*d0,unsigned char*operates,int oplen,unsigned long size,float*out,float*d1,float*d2,float*d3)
    {
        if (oplen>MAX_TRANSFORM) return INVALID_ARGUMENT;
        __set_op_transform(operates,oplen);
        
        dim3 gridDim;
        int dsect=(size+CUDALines-1)/CUDALines;
        gridDim.y=(dsect+1023)/1024; if (dsect>=1024) gridDim.x=1024; else gridDim.x=dsect;
        gridDim.z=(gridDim.y+1023)/1024; if (gridDim.y>=1024) gridDim.y=1024;

        sprintf(custom,"(%d,%d,%d,%d)",gridDim.x,gridDim.y,gridDim.z,CUDAAligns);
        ktransform<<<gridDim,CUDALines>>>(d0,d1,d2,d3,out,size,oplen,0);
        CUDAASSERT(hipDeviceSynchronize());
        return 0;
    }
	DLL extern int transform2(float*d0,unsigned char*operates,int oplen,unsigned long size,float*out,float*out2,float*d1,float*d2,float*d3)
	{
		if (oplen>MAX_TRANSFORM) return INVALID_ARGUMENT;
		__set_op_transform(operates,oplen);
		//Scan how many variables needed
		float*darr[4]={d0,d1,d2,d3};
		int largest=0;
		for (int p=0;p<oplen;p++)
			if (operates[p]==5) p+=4;
			else if (operates[p]<5)
				if (operates[p]>largest) largest=operates[p];
		float*deviced[4];
		for (int i=0;i<largest;i++)
			CUDAASSERT(hipMalloc(&deviced[i],CUDAMemory));
		float*deviceout,*deviceout2;
		CUDAASSERT(hipMalloc(&deviceout,CUDAMemory));
		CUDAASSERT(hipMalloc(&deviceout2,CUDAMemory));
		int sect=CUDAMemory/sizeof(float)/(largest==0?1:largest);
		if (sect>1024*256*CUDAAligns) sect=1024*256*CUDAAligns;
		int dsect=(sect+CUDALines-1)/CUDALines;
		for (unsigned long long ps=0;ps<size;ps+=sect)
		{
			for (int i=0;i<largest;i++)
				CUDAASSERT(hipMemcpy(deviced[i],&darr[i][ps],((size-ps>=sect)?sect:(size-ps))*sizeof(float),hipMemcpyHostToDevice));
			dim3 gridDim;

			gridDim.y=(dsect+1023)/1024; if (dsect>=1024) gridDim.x=1024; else gridDim.x=dsect;
			gridDim.z=(gridDim.y+1023)/1024; if (gridDim.y>=1024) gridDim.y=1024;
			sprintf(custom,"(%d,%d,%d,%d)",gridDim.x,gridDim.y,gridDim.z,CUDAAligns);
			ktransform2<<<gridDim,CUDALines>>>(deviced[0],deviced[1],deviced[2],deviced[3],deviceout,deviceout2,(size-ps>=sect)?sect:(size-ps),oplen,ps);
			CUDAASSERT(hipMemcpy(&out[ps],deviceout,((size-ps>=sect)?sect:(size-ps))*sizeof(float),hipMemcpyDeviceToHost));
			CUDAASSERT(hipMemcpy(&out2[ps],deviceout2,((size-ps>=sect)?sect:(size-ps))*sizeof(float),hipMemcpyDeviceToHost));
		}
		for (int i=0;i<largest;i++)
			CUDAASSERT(hipFree(deviced[i]));
		CUDAASSERT(hipFree(deviceout));
		CUDAASSERT(hipFree(deviceout2));
		return 0;
	}
    DLL extern int transformgpu2(float*d0,unsigned char*operates,int oplen,unsigned long size,float*out,float*out2,float*d1,float*d2,float*d3)
    {
        if (oplen>MAX_TRANSFORM) return INVALID_ARGUMENT;
        __set_op_transform(operates,oplen);

        dim3 gridDim;
        int dsect=(size+CUDALines-1)/CUDALines;
        gridDim.y=(dsect+1023)/1024; if (dsect>=1024) gridDim.x=1024; else gridDim.x=dsect;
        gridDim.z=(gridDim.y+1023)/1024; if (gridDim.y>=1024) gridDim.y=1024;

        sprintf(custom,"(%d,%d,%d,%d)",gridDim.x,gridDim.y,gridDim.z,CUDAAligns);
        ktransform2<<<gridDim,CUDALines>>>(d0,d1,d2,d3,out,out2,size,oplen,0);
        CUDAASSERT(hipDeviceSynchronize());
        return 0;
    }
}

